#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include "pre_cuda.h"
#include "opencv2/opencv.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/types_c.h"


inline int divUp(int a, int b) 
{
    assert(b > 0);
    return ceil((float) a / b);
};

__global__ void resize_op(uint8_t* src, uint8_t* dst,
                            float scale_x, float scale_y, int src_width, 
                            int src_height, int out_width, int out_height) {

    const int dst_x     = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_y     = blockIdx.y * blockDim.y + threadIdx.y;

    if ((dst_x < out_width) && (dst_y < out_height))
    {
        //y coordinate
        // 原图的y坐标, +0.5到像素坐标中心，否则是像素的左上角
        float fy = (float)((dst_y + 0.5f) * scale_y - 0.5f);
        // 左上角的y, 向下取整
        int   sy = floor(fy);
        fy -= sy;
        //  防止越界
        sy = max(0, min(sy, src_height - 2));
        //row pointers
        // sy,sx*3--BGR---BGR--BGR--sy,(sx+1)*3+1
        // ----------------------------------
        // sy+1,sx*3-BGR---BGR---BGR--sy+1,(sx+1)*3+1

        const uint8_t *aPtr = src + sy * src_width * 3;     //start of upper row
        const uint8_t *bPtr = src + (sy + 1) * src_width * 3; //start of lower row
        //compute source data position and weight for [x0] components
            float fx = (float)((dst_x + 0.5f) * scale_x - 0.5f);
            int   sx = floor(fx);
            fx -= sx;
            fx *= ((sx >= 0) && (sx < src_width - 1));
            sx = max(0, min(sx, src_width - 2));
        
        uint32_t sp = sx * 3;
        uint32_t sp_right = (sx + 1) * 3;
        uint32_t dp = dst_y * out_width * 3 + dst_x * 3;
        for (int i = 0; i < 3; i++) {
            dst[dp + i]
                = uint8_t((1.0f - fx) * (aPtr[sp + i] * (1.0f - fy) + bPtr[sp + i] * fy)
                            + fx * (aPtr[sp_right + i] * (1.0f - fy) + bPtr[sp_right + i] * fy));
        }

    }
}


__device__ inline void copymakeborder_op(
    uint8_t* src, uint8_t* dst,
    int top, int left, int out_width, 
    int out_height, uint8_t border_value
) {

}

namespace cudapre {

__global__ void copymakeborder_kernel(
                uint8_t *image, 
                uint8_t* out_image,
                uint32_t in_width, 
                uint32_t in_height,
                uint32_t out_width,
                uint32_t out_height, 
                int top,
                int left
) {
    
    const int src_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int src_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (src_x < in_width && src_y < in_height) {
        uint8_t* out_ptr = out_image + (src_y + top) * out_width * 3 + (src_x + left) * 3;
        uint8_t* in_ptr = image + src_y * in_width * 3 + src_x * 3;
        out_ptr[0] = in_ptr[0];
        out_ptr[1] = in_ptr[1];
        out_ptr[2] = in_ptr[2];
    }

}


// const nvcv::Tensor &inTensor, uint32_t batchSize, int inputLayerWidth, int inputLayerHeight,
                // hipStream_t stream, const nvcv::Tensor &outTensor

void gpu_resize(uint8_t *image, 
                uint8_t* outImage,
                hipStream_t stream,
                uint32_t src_width, 
                uint32_t src_height,
                uint32_t out_width,
                uint32_t out_height) 
{
    float scale_x = ((float)src_width) / out_width;
    float scale_y = ((float)src_height) / out_height;
    printf("scale_x: %f, scale_y: %f\n", scale_x, scale_y);


    const int batch_size = 1;
    const int THREADS_PER_BLOCK = 256; //256?  64?
    const int BLOCK_WIDTH       = 8;   //as in 32x4 or 32x8.  16x8 and 16x16 are also viable

    const dim3 blockSize(BLOCK_WIDTH, THREADS_PER_BLOCK / BLOCK_WIDTH, 1);
    const dim3 gridSize(divUp(out_width, blockSize.x), divUp(out_height, blockSize.y), batch_size);

    resize_op<<<gridSize, blockSize, 0, stream>>>(image, outImage, 
                                                scale_x, scale_y, src_width, 
                                                src_height, out_width, out_height);

    CHECK_RUN();
    // CHECK_CUDA_ERROR(hipMalloc(&indata, nums));
    // hipMemcpyAsync()
}


void copymakeborder(uint8_t *image, 
                uint8_t* outImage,
                hipStream_t stream,
                uint32_t in_width, 
                uint32_t in_height,
                uint32_t out_width,
                uint32_t out_height)
{
    int top = std::round(float(out_height - in_height) / 2 - 0.1f);
    int left = std::round(float(out_width - in_width) / 2 - 0.1f);

    const int batch_size = 1;
    const int THREADS_PER_BLOCK = 256; //256?  64?
    const int BLOCK_WIDTH       = 8;   //as in 32x4 or 32x8.  16x8 and 16x16 are also viable

    const dim3 blockSize(BLOCK_WIDTH, THREADS_PER_BLOCK / BLOCK_WIDTH, 1);
    const dim3 gridSize(divUp(in_width, blockSize.x), divUp(in_height, blockSize.y), batch_size);

    copymakeborder_kernel<<<gridSize, blockSize, 0, stream>>>
                (image, outImage, in_width, in_height, out_width, out_height, top, left);
    CHECK_RUN();

}


void cpu_resize(uint8_t* src, 
                uint8_t* dst,
                uint32_t src_width, 
                uint32_t src_height,
                uint32_t out_width,
                uint32_t out_height) {
    
    float scale_x = ((float)src_width) / out_width;
    float scale_y = ((float)src_height) / out_height;
    // printf("scale_x: %f, scale_y:%f\n", scale_x, scale_y);

    for (int dst_y = 0; dst_y < out_height; dst_y++) {
        for (int dst_x = 0; dst_x < out_width; dst_x++) {

            //float space for weighted addition
            // using work_type = cuda::ConvertBaseTypeTo<float, uint8_t>;

            //y coordinate
            // 原图的y坐标, +0.5到像素坐标中心，否则是像素的左上角
            double fy = double((dst_y + 0.5f) * scale_y - 0.5f);
            // 左上角的y, 向下取整
            int   top_y = std::round(fy);
            fy -= top_y;
            //  防止越界
            top_y = max(0, min(top_y, src_height - 2));

            //row pointers
            // top_y,left_y---BGR--BGR--BGR--top_y,left_x+1
            // ----------------------------------
            // top_y+1,left_y---BGR--BGR--BGR--top_y+1,left_x+1

            // BGRBGR

            const uint8_t *aPtr = src + top_y * src_width * 3;     //start of upper row
            const uint8_t *bPtr = src + (top_y + 1) * src_width * 3; //start of lower row

            //compute source data position and weight for [x0] components
                double fx = double((dst_x + 0.5f) * scale_x - 0.5f);
                int   left_x = std::round(fx);
                fx -= left_x;
                // fx *= ((left_x >= 0) && (left_x < src_width - 1));
                left_x = max(0, min(left_x, src_width - 2));
            
            uint32_t sp_left = left_x * 3;
            uint32_t sp_right = (left_x + 1) * 3;
            uint32_t dp = dst_y * out_width * 3 + dst_x * 3;
            for (int i = 0; i < 3; i++) {
                dst[dp + i]
                    = uint8_t((1.0f - fx) * (aPtr[sp_left + i] * (1.0f - fy) + bPtr[sp_left + i] * fy)
                             + fx * (aPtr[sp_right + i] * (1.0f - fy) + bPtr[sp_right + i] * fy));
            }
        }
    }
}


void cpu_copymakeborder(uint8_t* src, 
                uint8_t* dst,
                uint32_t in_width,
                uint32_t in_height,
                uint32_t out_width,
                uint32_t out_height,
                uint8_t border_value) 
{
    int top = std::round(float(out_height - in_height) / 2 - 0.1f);
    int left = std::round(float(out_width - in_width) / 2 - 0.1f);
    
    // dst默认初始化为114的值, 所以空闲位置不用考虑
    for (int i = 0; i < in_height; i++) {
        uint8_t* dst_ptr = dst + (i + top) * out_width * 3;
        uint8_t* src_ptr = src + i * in_width * 3;
        for (int j = 0; j < in_width; j++) {
            dst_ptr[(j + left) * 3] = src_ptr[j * 3];
            dst_ptr[(j + left) * 3 + 1] = src_ptr[j * 3 + 1];
            dst_ptr[(j + left) * 3 + 2] = src_ptr[j * 3 + 2];
        }
    }
}


void cpu_letterbox(uint8_t* src, 
                uint8_t* dst,
                uint32_t src_width, 
                uint32_t src_height,
                uint32_t out_width,
                uint32_t out_height
) {
    // cpu_resize(src, dst, src_width, src_height, out_width, out_height);


}

};

